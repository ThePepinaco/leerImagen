#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"



#define FILTER_SIZE 21
#define FILTER_RADIUS (FILTER_SIZE / 2)
#define SIGMA 100.0f
// Tamaño y sigma del kernel


std::vector<std::vector<float>> createGaussianBlurKernel(int size, float sigma) {
    std::vector<std::vector<float>> kernel(size, std::vector<float>(size, 0.0f));
    float sum = 0.0f;
    int half = size / 2;

    for (int i = -half; i <= half; ++i) {
        for (int j = -half; j <= half; ++j) {
            float value = exp(-(i * i + j * j) / (2 * sigma * sigma));
            kernel[i + half][j + half] = value;
            sum += value;
        }
    }
    // Normalizar el kernel
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            kernel[i][j] /= sum;
        }
    }

    return kernel;
}

void gaussianBlurCPU(const unsigned char* inputImage, unsigned char* outputImage, int width, int height, const std::vector<std::vector<float>>& kernel) {
    
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            float sum = 0.0f;
            float value = 0.0f;

            for (int i = -FILTER_RADIUS; i <= FILTER_RADIUS; ++i) {
                for (int j = -FILTER_RADIUS; j <= FILTER_RADIUS; ++j) {
                    int offsetX = x + j;
                    int offsetY = y + i;

                    if (offsetX >= 0 && offsetX < width && offsetY >= 0 && offsetY < height) {
                        float weight = kernel[i + FILTER_RADIUS][j + FILTER_RADIUS];
                        value += weight * inputImage[offsetY * width + offsetX];
                        sum += weight;
                    }
                }
            }

            outputImage[y * width + x] = static_cast<unsigned char>(value / sum);
        }
    }
}

__global__ void gaussianBlurKernelCUDA(const unsigned char* inputImage, unsigned char* outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float sum = 0.0f;
        float value = 0.0f;

        // aplicar el filtro usando las posiciones x e y
        for (int i = -FILTER_RADIUS; i <= FILTER_RADIUS; ++i) {
            for (int j = -FILTER_RADIUS; j <= FILTER_RADIUS; ++j) {
                int offsetX = x + j;
                int offsetY = y + i;

                // verificar si no estamos en los bordes
                if (offsetX >= 0 && offsetX < width && offsetY >= 0 && offsetY < height) {
                    float weight = expf(-(i * i + j * j) / (2.0f * FILTER_RADIUS * FILTER_RADIUS));
                    sum += weight;
                    value += weight * inputImage[offsetY * width + offsetX];
                }
            }
        }

        outputImage[y * width + x] = static_cast<unsigned char>(value / sum);
    }
}

int main() {
    int width, height, channels;

    unsigned char* image = stbi_load("1.jpg", &width, &height, &channels, 0);

    if (!image) {
        std::cerr << "No se pudo abrir la imagen." << std::endl;
        return 1;
    }
    std::vector<std::vector<float>> gaussianBlurKernel = createGaussianBlurKernel(FILTER_SIZE, SIGMA);
    unsigned char* grayscaleImage = new unsigned char[width * height];

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            // Calculate grayscale value
            int index = (y * width + x) * channels;
            int r = image[index];
            int g = image[index + 1];
            int b = image[index + 2];
            grayscaleImage[y * width + x] = (unsigned char)((r + g + b) / 3);
        }
    }

    size_t imageSize = width * height * sizeof(unsigned char);





    unsigned char* d_inputImage, * d_outputImage;
    hipMalloc(&d_inputImage, imageSize);
    hipMalloc(&d_outputImage, imageSize);

    hipMemcpy(d_inputImage, grayscaleImage, imageSize, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    auto start = std::chrono::high_resolution_clock::now();

    gaussianBlurKernelCUDA << <gridDim, blockDim >> > (d_inputImage, d_outputImage, width, height);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;

    std::cout << "Tiempo de ejecución: " << duration.count() << "ms  GPU --- FILTRO GAUSS BLUR" << std::endl;

    unsigned char* blurredImage = new unsigned char[width * height];
    hipMemcpy(blurredImage, d_outputImage, imageSize, hipMemcpyDeviceToHost);

    stbi_write_png("FiltroGaussBlurGPU.jpg", width, height, 1, blurredImage, width);
    






    unsigned char* blurredImage1 = new unsigned char[width * height];

    auto start1 = std::chrono::high_resolution_clock::now();

    // Aplicar el filtro de desenfoque gaussiano en la CPU
    gaussianBlurCPU(grayscaleImage, blurredImage1, width, height, gaussianBlurKernel);

    auto end1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration1 = end1 - start1;

    std::cout << "Tiempo de ejecución: " << duration1.count() << "ms  CPU --- FILTRO GAUSS BLUR" << std::endl;

    // Guardar la imagen desenfocada
    stbi_write_png("FiltroGaussBlurCPU.jpg", width, height, 1, blurredImage1, width);




    delete[] grayscaleImage;
    delete[] blurredImage;
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    return 0;
}
