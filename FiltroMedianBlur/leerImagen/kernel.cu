#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <chrono>
#include <algorithm>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define KERNEL_SIZE 3
#define KERNEL_RADIUS (KERNEL_SIZE / 2)

int median(const std::vector<int>& v) {
    std::vector<int> sorted = v;
    std::sort(sorted.begin(), sorted.end());
    return sorted[v.size() / 2];
}

void applyMedianFilter(const unsigned char* input, unsigned char* output, int width, int height) {
    
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            std::vector<int> neighborhood;
            for (int ky = -KERNEL_RADIUS; ky <= KERNEL_RADIUS; ky++) {
                for (int kx = -KERNEL_RADIUS; kx <= KERNEL_RADIUS; kx++) {
                    int nx = std::min(std::max(x + kx, 0), width - 1);
                    int ny = std::min(std::max(y + ky, 0), height - 1);
                    neighborhood.push_back(input[ny * width + nx]);
                }
            }

            std::sort(neighborhood.begin(), neighborhood.end());
            output[y * width + x] = neighborhood[neighborhood.size() / 2];
        }
    }
}

int main() {
    int width, height, channels;

    unsigned char* image = stbi_load("images.jpg", &width, &height, &channels, 0);

    if (!image) {
        std::cerr << "No se pudo abrir la imagen." << std::endl;
        return 1;
    }

    unsigned char* grayscaleImage = new unsigned char[width * height];

    // Convertir a escala de grises
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            // Calculate grayscale value
            int index = (y * width + x) * channels;
            int r = image[index];
            int g = image[index + 1];
            int b = image[index + 2];
            grayscaleImage[y * width + x] = (unsigned char)((r + g + b) / 3);
        }
    }

    unsigned char* filteredImage = new unsigned char[width * height];

    // Aplicar filtro de mediana
    auto start1 = std::chrono::high_resolution_clock::now();
    applyMedianFilter(grayscaleImage, filteredImage, width, height);
    auto end1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration1 = end1 - start1;
    std::cout << "Tiempo de ejecución: " << duration1.count() << "ms  CPU --- FILTRO MEDIAN BLUR" << std::endl;
    

    // Guardar la imagen filtrada
    stbi_write_png("FiltroMedianBlurCPU.jpg", width, height, 1, filteredImage, width);

    // Liberar memoria
    stbi_image_free(image);
    delete[] grayscaleImage;
    delete[] filteredImage;

    return 0;
}
